/* $Id:  $        */
/*--------------------------------------------------------------------*/
/*;  Copyright (C) 2014                                               */
/*;  Associated Universities, Inc. Washington DC, USA.                */
/*;                                                                   */
/*;  This program is free software; you can redistribute it and/or    */
/*;  modify it under the terms of the GNU General Public License as   */
/*;  published by the Free Software Foundation; either version 2 of   */
/*;  the License, or (at your option) any later version.              */
/*;                                                                   */
/*;  This program is distributed in the hope that it will be useful,  */
/*;  but WITHOUT ANY WARRANTY; without even the implied warranty of   */
/*;  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the    */
/*;  GNU General Public License for more details.                     */
/*;                                                                   */
/*;  You should have received a copy of the GNU General Public        */
/*;  License along with this program; if not, write to the Free       */
/*;  Software Foundation, Inc., 675 Massachusetts Ave, Cambridge,     */
/*;  MA 02139, USA.                                                   */
/*;                                                                   */
/*;Correspondence about this software should be addressed as follows: */
/*;         Internet email: bcotton@nrao.edu.                         */
/*;         Postal address: William Cotton                            */
/*;                         National Radio Astronomy Observatory      */
/*;                         520 Edgemont Road                         */
/*;                         Charlottesville, VA 22903-2475 USA        */
/*--------------------------------------------------------------------*/
#ifndef OBITCUDAUTIL_H 
#define OBITCUDAUTIl_H 

#define IS_CUDA 1

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper for shared that are common to CUDA SDK samples

/*-------- Obit: Merx mollis mortibus nuper ------------------*/
/**
 * \file ObitCUDAUtil.h
 *
 * C callable CUDA GPU routines
 * Routines abort on error
 */

/*---------------Public functions---------------------------*/
#if HAVE_GPU==1  /* CUDA code */
/* Public: Set device */
/**
 * Assign a GPU
 * \param cuda_device GPU number to use
 */
extern "C"
void ObitCUDASetGPU (int cuda_device)
{
  checkCudaErrors(hipSetDevice(cuda_device));
} /* end ObitCUDASetGPU */

/**
 * Reset device 
 */
extern "C"
void ObitCUDAResetGPU ()
{
  hipDeviceReset();
} /* end ObitCUDAResetGPU */

/**
 * Synchronize the device
 * \param event  that to be waited for, not really used
 */
extern "C"
void ObitCUDADeviceSynchronize (int* event)
{
  hipDeviceSynchronize();
} /* end ObitCUDADeviceSynchronize */

/**
 * Create a processing stream
 * Native CUDA types
 * \return stream
 */
hipStream_t ObitCUDAStreamCreateCUDA ()
{
  hipStream_t out=NULL;
  checkCudaErrors(hipStreamCreate(&out));
  return out;
} /* end ObitCUDAStreamCreateCUDA */

/**
 * Destroy a stream
 * Native CUDA types
 * \param stream stream to destroy
 */
void ObitCUDAStreamDestroyCUDA (hipStream_t stream)
{
  hipStreamDestroy(stream);
} /* end ObitCUDAStreamDestroyCUDA */

/**
 * Create an event
 * Native CUDA types
 * \return event
 */
hipEvent_t ObitCUDAEventCreateCUDA ()
{
  hipEvent_t out=NULL;
  checkCudaErrors(hipEventCreate(&out));
  return out;
} /* end ObitCUDAEventCreateCUDA */

/**
 * Destroy an event
 * Native CUDA types
 * \param event to be destroyed
 */
void ObitCUDAEventDestroyCUDA (hipEvent_t event)
{
  hipEventDestroy(event);
} /* end ObitCUDAEventDestroyCUDA */

/**
 * Associate an event with a stream, waiting for completion
 * Native CUDA types
 * \param event  to wait for
 * \param stream stream
 */
void ObitCUDAEventRecordCUDA (hipEvent_t event, hipStream_t stream)
{
  hipEventRecord(event, stream);
} /* end ObitCUDAEventRecordCUDA */

/**
 * Wait for an event defined by ObitCUDAEventRecordCUDA
 * Native CUDA types
 * \param event  to wait for
 */
void ObitCUDAEventSynchronizeCUDA (hipEvent_t event)
{
  hipEventSynchronize(event);
} /* end ObitCUDAEventSynchronizeCUDA */

/**
 * Create a processing stream
 * \return stream as int*
 */
extern "C"
int* ObitCUDAStreamCreate ()
{
  int* out=NULL;
  checkCudaErrors(hipStreamCreate((hipStream_t*)&out));
  return out;
} /* end ObitCUDAStreamCreate */

/**
 * Destroy a stream
 * \param stream stream to destroy (as int*)
 */
extern "C"
void ObitCUDAStreamDestroy (int* stream)
{
  hipStreamDestroy((hipStream_t)stream);
} /* end ObitCUDAStreamDestroy */

/**
 * Create an event
 * \return event as int*
 */
extern "C"
int* ObitCUDAEventCreate ()
{
  int* out=NULL;
  checkCudaErrors(hipEventCreate((hipEvent_t*)&out));
  return out;
} /* end ObitCUDAEventCreate */

/**
 * Destroy an event
 * \param event to be destroyed (as int*)
 */
extern "C"
void ObitCUDAEventDestroy (int* event)
{
  hipEventDestroy((hipEvent_t)event);
} /* end ObitCUDAEventDestroy */

/**
 * Associate an event with a stream, waiting for completion
 * \param event  to wait for
 * \param stream stream
 */
extern "C"
void ObitCUDAEventRecord (int* event, int* stream)
{
  hipEventRecord((hipEvent_t)event, (hipStream_t)stream);
} /* end ObitCUDAEventRecord */

/**
 * Wait for an event defined by ObitCUDAEventRecord
 * \param event  to wait for
 */
extern "C"
void ObitCUDAEventSynchronize (int* event)
{
  hipEventSynchronize((hipEvent_t)event);
} /* end ObitCUDAEventSynchronize */

/**
 * Allocate locked host memory 
 * \param memsize  size in bytes
 * \return pointer to memory
 */
extern "C"
float* ObitCUDAUtilAllocHost (int memsize)
{
  float *out=NULL;
  checkCudaErrors(hipHostMalloc(&out, memsize));
  return out;
} /* end ObitCUDAUtilAllocHost */

/**
 * Deallocate locked host memory
 * \param host memory pointer to free
 */
extern "C"
void ObitCUDAUtilFreeHost (float *host)
{
  hipHostFree(host);
} /* end ObitCUDAUtilFreeHost */

/**
 * Allocate device memory 
 * \param  size in bytes
 * \return pointer to memory
 */
extern "C"
float* ObitCUDAUtilAllocGPU (int memsize)
{
  float *out=NULL;
  checkCudaErrors(hipMalloc(&out, memsize));
  return out;
} /* end ObitCUDAUtilAllocGPU */

/**
 * Deallocate Device memory
 * \param GPU memory pointer to free
 */
extern "C"
void ObitCUDAUtilFreeGPU (float *GPU)
{
  hipFree(GPU);
} /* end ObitCUDAUtilFreeGPU */

/**
 * Copy data from host to GPU memory
 * \param GPU      GPU memory
 * \param host     locked host memory
 * \param memsize  size in bytes
 * \param stream   If non-NULL then stream pointer
 */
extern "C"
void ObitCUDAUtilHost2GPU(float *GPU, float *host, int memsize, int* stream)
{
  if (stream!=NULL) {
    checkCudaErrors(hipMemcpyAsync(GPU, host, memsize, 
      hipMemcpyHostToDevice, (hipStream_t)stream));
  } else {
    checkCudaErrors(hipMemcpyAsync(GPU, host, memsize, hipMemcpyHostToDevice, 0));
  }
} /* end ObitCUDAUtilHost2GPU */

/**
 * Copy data from GPU to host memory
 * \param host     locked host memory
 * \param GPU      GPU memory
 * \param memsize  size in bytes
 * \param stream   If non-NULL then stream pointer
 */
extern "C"
void ObitCUDAUtilGPU2Host(float *host, float *GPU, int memsize, int* stream)
{
  if (stream!=NULL) {
    checkCudaErrors(hipMemcpyAsync(host,GPU,  memsize, 
      hipMemcpyDeviceToHost, (hipStream_t)stream));
  } else {
    checkCudaErrors(hipMemcpyAsync(host, GPU, memsize, hipMemcpyDeviceToHost, 0));
  }
} /* end ObitCUDAUtilGPU2Host */
#endif /* HAVE_GPU */


/**
 * Returns Obit magic blanking float value
 * This is adopted from AIPS and correcponds to the string 'INDE'
 * \return float magic value
 */
float CUDAMagicF (void)
{
  static union FBLANKequiv {
    char string[4];
    float fblank;
  } FBLANK;
  FBLANK.string[0] = 'I'; 
  FBLANK.string[1] = 'N'; 
  FBLANK.string[2] = 'D'; 
  FBLANK.string[3] = 'E'; 
  
  return FBLANK.fblank;
} /* end CUDAMagicF */
#endif /* OBITFCUDAUtil_H */ 

