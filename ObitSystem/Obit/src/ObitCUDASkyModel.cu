#include "hip/hip_runtime.h"
/* Still need 
2) Spectral index
4) Gaussian model 
5) Grid
*/
/* $Id: $        */
/*--------------------------------------------------------------------*/
/*;  Copyright (C) 2014                                               */
/*;  Associated Universities, Inc. Washington DC, USA.                */
/*;                                                                   */
/*;  This program is free software; you can redistribute it and/or    */
/*;  modify it under the terms of the GNU General Public License as   */
/*;  published by the Free Software Foundation; either version 2 of   */
/*;  the License, or (at your option) any later version.              */
/*;                                                                   */
/*;  This program is distributed in the hope that it will be useful,  */
/*;  but WITHOUT ANY WARRANTY; without even the implied warranty of   */
/*;  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the    */
/*;  GNU General Public License for more details.                     */
/*;                                                                   */
/*;  You should have received a copy of the GNU General Public        */
/*;  License along with this program; if not, write to the Free       */
/*;  Software Foundation, Inc., 675 Massachusetts Ave, Cambridge,     */
/*;  MA 02139, USA.                                                   */
/*;                                                                   */
/*;Correspondence about this software should be addressed as follows: */
/*;         Internet email: bcotton@nrao.edu.                         */
/*;         Postal address: William Cotton                            */
/*;                         National Radio Astronomy Observatory      */
/*;                         520 Edgemont Road                         */
/*;                         Charlottesville, VA 22903-2475 USA        */
/*--------------------------------------------------------------------*/

/*#include "ObitCUDASkyModel.h"*/
#include "ObitCUDASkyModelInfoDef.h"
/*----------------Obit: Merx mollis mortibus nuper ------------------*/
/**
 * \file ObitCUDASkyModel.cu
 * Primitive CUDA routines
 * Portions of the class are in CUDA and are only implemented if the
 * compiler option -DHAVE_GPU=1 is used.  Some portions also need to 
 * variable IS_CUDA=1 to be set in the calling routines.
 */

/*--------------- CUDA setup, GPU kernal  ----------------*/
/* This is a CUDA routine */
#if HAVE_GPU==1  /* CUDA code */
#define IS_CUDA 1

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper for shared that are common to CUDA SDK samples

/* DEBUG */
float *h_debug, *d_debug;
__global__ void debugKernal( float* __restrict__ d_debug, GPUVisInfo* visInfo)
{
    int iprod      = threadIdx.x+256*blockIdx.y;     // product number
    // No more than actual number of products
    if (iprod>=visInfo->nprod) return;
    d_debug[0] = blockIdx.x;
    d_debug[1] = blockIdx.y;
    d_debug[2] = threadIdx.x;
    d_debug[3] = threadIdx.y;
} /* end debugKernel */
/**
 * Point DFT GPU kernal.
 * block = vis, threads in block = data product = channel/stokes/IF
 * does full model for one data product
 * \param  g_out      input data
 * \param  g_in       output data
 * \param  modelInfo  model information
 * \param  visInfo    visibility information
 */
__global__ void dftPointKernel(float* __restrict__ g_out, float* __restrict__ g_in, 
	   GPUModelInfo* modelInfo,  GPUVisInfo* visInfo)
{
    int lenvis     = visInfo->lenvis;
    int idx        = blockIdx.x * lenvis;        // beginning of a visibility
    int iprod      = threadIdx.x+256*blockIdx.y; // product number
    int nrparm     = visInfo->nrparm;
    float *FreqArr = visInfo->freqScale;
    int nModel     = modelInfo->nmodel;
    int modelSize  = modelInfo->size;
    float *Model   = modelInfo->model;
    int i, ichan, istok, iIF, ivis;
    int iMod = 0;
    float arg, amp, s, c, sumR, sumI; 
    float u, v, w;
    float freqFact;

    // No more than actual number of products
    if (iprod>=visInfo->nprod) return;

    // get channel,stokes, IF from data product
    ichan = (iprod / visInfo->incf)  % visInfo->nchan;
    istok = (iprod / visInfo->incs)  % visInfo->nstok;
    iIF   = (iprod / visInfo->incif) % visInfo->nIF;

   // This one desired?
    if ((ichan<visInfo->chanb) || (ichan>visInfo->chane) ||
        (istok<visInfo->stokb) || (istok>visInfo->stoke) ||
	(iIF<visInfo->IFb)     || (iIF>visInfo->IFe)) return;

    // frequency scaling factor	  
    freqFact = FreqArr[ichan*visInfo->kincf+iIF*visInfo->kincif];
    // copy random parameters if first channel
    if (ichan==visInfo->chanb) {
       for (i=0; i<nrparm; i++) g_out[idx+i] = g_in[idx+i];
    }
 
    // real part of vis
    ivis = idx + nrparm + iprod*3;
 
  // bail if weight non positive, doesn't really matter
    //if (g_in[ivis+2]<=0.0) return;

   // get scaled u,v,w factors
    u = g_in[idx+visInfo->ilocu];  // this seems faster???
    v = g_in[idx+visInfo->ilocv];
    w = g_in[idx+visInfo->ilocw];
    u *= freqFact;
    v *= freqFact;
    w *= freqFact;
    sumR = sumI = 0.0;
    // model = flux, x,y,z factors

    for (int i=0; i<nModel; i++) {
	amp = Model[iMod];
 	arg = u*Model[iMod+1] + v*Model[iMod+2] + w*Model[iMod+3];
	__sincosf(arg, &s, &c);
        sumR += amp * c;
        sumI += amp * s;
        iMod += modelSize;
    } // end loop over model comps
    g_out[ivis]   = g_in[ivis]   - sumR;
    g_out[ivis+1] = g_in[ivis+1] - sumI;
    g_out[ivis+2] = g_in[ivis+2];
 } // end dftPointKernel
#endif /* HAVE_GPU */

#if HAVE_GPU==1  /* CUDA code */
static void DFTprocessWithStreams(int streams_used, int nvis, 
            GPUModelInfo* h_modelInfo, GPUModelInfo* d_modelInfo, 
            GPUVisInfo* h_visInfo, GPUVisInfo* d_visInfo, 
            hipStream_t* stream, hipEvent_t* cycleDone,
            float *h_data_source, float *h_data_sink, 
            float *d_data_in[], float *d_data_out[], int prtLv);
#endif /* HAVE_GPU */

/*----------------------Public functions---------------------------*/
/**
 * Initialize an ObitCUDASkyModel 
 * Currently only DFT point supported
 * \param gpuInfo    processing info
 * \param visInfo    visibility info
 * \param modelInfo  model info
 */
extern "C"
void ObitCUDASkyModelDFTInit (GPUModelInfo *gpuInfo, GPUVisInfo *visInfo, GPUModelInfo *modelInfo)
{
} /* end  ObitCUDASkyModelDFTInit */


/**
 * Setup model for ObitCUDASkyModel 
 * Currently only DFT point supported
 * \param gpuInfo    processing info
 * \param visInfo    visibility info
 * \param modelInfo  model info
 */
extern "C"
void ObitCUDASkyModelDFTSetMod (GPUModelInfo *gpuInfo, GPUVisInfo *visInfo, GPUModelInfo *modelInfo)
{
#if HAVE_GPU==1  /* CUDA code */
#endif /* HAVE_GPU */
    return;
} /* end ObitCUDASkyModelDFTSetMod */

/**
 * Calculate an ObitCUDASkyModel 
 * Currently only DFT point supported
 * \param gpuInfo    processing info
 * \param visInfo    visibility info
 * \param modelInfo  model info
 */
extern "C"
void ObitCUDASkyModelDFTCalc (GPUInfo *gpuInfo, GPUVisInfo *visInfo, GPUModelInfo *modelInfo)
{
#if HAVE_GPU==1  /* CUDA code */
   // Put available fast memory in L1 cache - no apparent effect
   checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(dftPointKernel), hipFuncCachePreferL1));

   /* Process with streams */
   DFTprocessWithStreams(gpuInfo->nstream, gpuInfo->nvis, modelInfo, gpuInfo->d_modelInfo, 
                        gpuInfo->h_visInfo, gpuInfo->d_visInfo, 
                        (hipStream_t *)gpuInfo->stream, (hipEvent_t *)gpuInfo->cycleDone, 
                        gpuInfo->h_data, gpuInfo->h_data, gpuInfo->d_data_in, gpuInfo->d_data_out, 0);
#endif /* HAVE_GPU */
    return;
} /* end ObitCUDASkyModelDFTCalc */


/**
 * Shutdown  ObitCUDASkyModel 
 * Currently only DFT point supported
 * \param gpuInfo    processing info
 * \param visInfo    visibility info
 * \param modelInfo  model info
 */
extern "C"
void ObitCUDASkyModelDFTShutdown (GPUInfo *gpuInfo, GPUVisInfo *visInfo, GPUModelInfo *modelInfo)
{

#if HAVE_GPU==1  /* CUDA code */
#endif /* HAVE_GPU */

} /*end ObitCUDASkyModelDFTShutdown */
#if HAVE_GPU==1  /* CUDA code */
/**
 * Calculates DFT model using a GPU.
 * Multiple streams are used to overlap I/O and computation,
 * each call divides the data into streams_used pieces.
 * \param  streams_used  Number of streams to use
 * \param  nvis          Number of visibilities
 * \param  nmodel        Number of sky model components
 * \param  h_modelInfo   Host resident modelInfo
 * \param  d_modelInfo   GPU resident modelInfo
 * \param  h_visInfo     Host resident visInfo
 * \param  d_visInfo     GPU resident visInfo
 * \param  stream        GPU stream array [streams_used]
 * \param  cycleDone     GPU event array [streams_used]
 * \param  h_data_source Host resident input buffer, should be locked
 * \param  h_data_sink   Host resident output buffer, should be locked
 *                       may be h_data_source
 * \param  d_data_in     GPU resident input data buffer (nvis/streams_used)
 * \param  d_data_out    GPU resident output data buffer (nvis/streams_used)
 * \param  prtLv         Print level, 5=>much directly printed
 */
static void DFTprocessWithStreams(int streams_used, int nvis, 
            GPUModelInfo* h_modelInfo, GPUModelInfo* d_modelInfo, 
            GPUVisInfo* h_visInfo, GPUVisInfo* d_visInfo, 
            hipStream_t* stream, hipEvent_t* cycleDone,
            float *h_data_source, float *h_data_sink, 
            float *d_data_in[], float *d_data_out[], int prtLv)
{

    int  last_stream, current_stream = 0;
    int npass = streams_used;
    int nvisPass = (nvis+npass+1)/npass;  // round up
    int lenvis = h_visInfo->lenvis;
    int off, nprod, dovis;
    int memsize = (lenvis*nvisPass)*sizeof(float);
    dim3 numBlocks, thPerBlock;

    // DEBUG
    //int ms = 10000*sizeof(float);
    //checkCudaErrors(hipMalloc(&d_debug, ms));
    //checkCudaErrors(hipHostMalloc(&h_debug, ms));
    //h_debug[0]=999.999;
    //checkCudaErrors(hipMemcpyAsync(d_debug,h_debug,  ms, hipMemcpyHostToDevice,0));

    // Number opf data products
    nprod = h_visInfo->nchan * h_visInfo->nstok * h_visInfo->nIF;

    if (prtLv>=5) printf ("Start\n");
 
    // Do processing in a loop
    //
    // Note: All memory commands are processed in the order  they are issued,
    // independent of the stream they are enqueued in. Hence the pattern by
    // which the copy and kernel commands are enqueued in the stream
    // has an influence on the achieved overlap.


    // Upload first frame
    if (prtLv>=5) printf ("upload current_stream %d off %d\n",0,0);
    checkCudaErrors(hipMemcpyAsync(d_data_in[0],
			  	   &h_data_source[0],
				   memsize,
				   hipMemcpyHostToDevice,
				   stream[0]));
    //?hipEventSynchronize(cycleDone[0]);

    for (int i=0; i<npass; ++i) {
        int next_stream = (current_stream + 1) % streams_used;
	int prev_stream = current_stream - 1;
	if (prev_stream<0) prev_stream = streams_used-1;
	off = next_stream*lenvis*nvisPass;  /* Offset in data buffers */
  	if (prtLv>=5) printf ("\n\nLoop %d prev %d current %d next %d\n",i, prev_stream,current_stream,next_stream );

	// Upload next frame
	if (prtLv>=5) printf ("upload next_stream %d off %d\n",next_stream,off);
	checkCudaErrors(hipMemcpyAsync(d_data_in[next_stream],
			    &h_data_source[off],
                            memsize,
                            hipMemcpyHostToDevice,
                            stream[next_stream]));

	// Ensure that processing and copying of the previous cycle has finished
	if (i>0) {
	  off = prev_stream*lenvis*nvisPass;  /* Offset in data buffers */
	  if (prtLv>=5) printf ("download prev_stream %d off %d\n",prev_stream,off);
	  checkCudaErrors(hipMemcpyAsync(&h_data_sink[off],
					  d_data_out[prev_stream],
					  memsize,
					  hipMemcpyDeviceToHost,
					  stream[prev_stream]));
	  if (prtLv>=5) printf ("sync prev_stream %d loop %d\n",prev_stream, i);
	  hipEventSynchronize(cycleDone[prev_stream]);
	}

        // Process current
	if (prtLv>=5) printf ("Process, nvis, %d nch %d stream %d \n",nvisPass, h_visInfo->nchan, current_stream);
	// make sure to do all visibilities
	if (i==npass-1) dovis = nvis-i*nvisPass;
	else            dovis = nvisPass;

	// package work
	numBlocks.x  = dovis; numBlocks.y = (nprod+127)/128;
	thPerBlock.x = 128;   thPerBlock.y = 1;
	//debugKernal<<<numBlocks, thPerBlock, 0, stream[current_stream]>>>(d_debug, d_visInfo);
	dftPointKernel<<<numBlocks, thPerBlock, 0, stream[current_stream]>>>(
            d_data_out[current_stream],
            d_data_in[current_stream],
            d_modelInfo, d_visInfo);

	// make sure previous frame done
	if (i>0) hipEventSynchronize(cycleDone[prev_stream]);

	last_stream = current_stream;
        current_stream = next_stream;
    } /* end loop */

    /* Data from last pass */
    if (prtLv>=5) printf ("sync last_stream %d \n",last_stream);
    hipEventSynchronize(cycleDone[last_stream]);
    
    // last piece may be uneven size
    dovis   = nvis-nvisPass*(npass-1);
    memsize = (lenvis*dovis)*sizeof(float);
    off = last_stream*lenvis*nvisPass;  /* Offset in data buffers */
    if (prtLv>=5) printf ("download last_stream %d off %d\n",last_stream,off);
    checkCudaErrors(hipMemcpyAsync(&h_data_sink[off],
				    d_data_out[last_stream],
				    memsize,
				    hipMemcpyDeviceToHost,
				    stream[last_stream]));
    if (prtLv>=5) printf ("Finish\n");
    hipDeviceSynchronize();

    // DEBUG
    //checkCudaErrors(hipMemcpyAsync(h_debug, d_debug, ms, hipMemcpyDeviceToHost,0));
    //hipHostFree(h_debug);
    //hipFree(d_debug);		 

    return;
} /* end processWithStreams */
#endif /* HAVE_GPU */

